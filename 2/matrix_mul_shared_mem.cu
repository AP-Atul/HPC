#include "hip/hip_runtime.h"
%%cu
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#define row1 5
#define col1 4
#define row2 4
#define col2 5
#define RANGE 10
using namespace std;

__global__ void matrixMulSharedMemory(long * deviceA, long * deviceB, long * deviceC)
{
    int x = blockIdx.x;
    int y = blockIdx.y;
    __shared__ long answer[col1];
    int tid = threadIdx.x;

    deviceC[col2 * y + x] = 0;
    answer[tid] = deviceA[col1 * y + tid] * deviceB[col2 * tid + x];

    __syncthreads();

    for(int i = 0; i < col1; i++) deviceC[col2 * y + x] += answer[i];
}

void printMatrix(long a[row1][col2]){
  for(int i = 0;  i < row1; i++){
    for(int j = 0 ; j < col2; j++)
      cout << a[i][j] << " ";
    cout << endl;
  }
}

//# host multiplication
void mulCPU(long hostA[row1][col1], long hostB[row2][col2], long actual[row1][col2]){
    for(int i = 0; i < row1; ++i) 
      for(int j = 0; j < col2; ++j) {
        actual[i][j] = 0;
        for(int k = 0; k < col1; ++k)
          actual[i][j] += hostA[i][k] * hostB[k][j];
      }
}


int main()
{
  long a[row1][col1];
  long b[row2][col2];
  long c[row1][col2];
  bool print = 1;

  long * deviceA, * deviceB, * deviceC;

  for(int i = 0; i < row1; i++)
  {
      for(int j = 0; j < col1; j++)
          {
              a[i][j] = rand() % RANGE;
              b[i][j] = rand() % RANGE;
          }
  }

  //# ************* GPU MUL *******************// 
  hipMalloc((void **)&deviceA, row1 * col1 * sizeof(long));
  hipMalloc((void **)&deviceB, row2 * col2 * sizeof(long));
  hipMalloc((void **)&deviceC, row1 * col2 * sizeof(long));

  hipMemcpy(deviceA, a, row1 * col1 * sizeof(long), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, b, row2 * col2 * sizeof(long), hipMemcpyHostToDevice);

  dim3 grid(col2,row1);
  clock_t start = clock();
  matrixMulSharedMemory<<<grid,col1>>>(deviceA, deviceB, deviceC);
  double time = clock() - start;

  hipMemcpy(c, deviceC, row1 * col2 * sizeof(long), hipMemcpyDeviceToHost);
  printf("Time taken (parallel) :: %lf", time / CLOCKS_PER_SEC);

  //# ***************** CPU MUL ****************//
  long actual[row1][col2];
  
  start = clock();
  mulCPU(a, b, actual);
  time = clock() - start;

  printf("\nTime taken (serial) :: %lf", time / CLOCKS_PER_SEC);

  if(print){
    printf("\n\nCPU matrix :: \n");
    printMatrix(actual);
  
    printf("\n\nGPU matrix :: \n");
    printMatrix(c);
  }
 
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  return 0;
}