#include "hip/hip_runtime.h"
%%cu
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#define row1 200
#define col1 300
#define row2 300
#define col2 200
#define RANGE 1000

__global__ void matrixMulSharedMemory(int * deviceA, int * deviceB, int * deviceC)
{
    int x = blockIdx.x;
    int y = blockIdx.y;
    __shared__ int answer[col1];
    int tid = threadIdx.x;

    deviceC[col2 * y + x] = 0;
    answer[tid] = deviceA[col1 * y + tid] * deviceB[col2 * tid + x];

    __syncthreads();

    for(int i = 0;i < col1; i++) deviceC[col2 * y + x] += answer[i];
}

//# host multiplication
void mulCPU(int hostA[row1][col1], int hostB[row2][col2], int actual[row1][col2]){
    for(int i = 0; i < row1; ++i) 
      for(int j = 0; j < col2; ++j) 
        for(int k = 0; k < col1; ++k) 
          actual[i][j] += hostA[i][k] * hostB[k][j];
}

int main()
{
  int a[row1][col1];
  int b[row2][col2];
  int c[row1][col2];

  int * deviceA, * deviceB, * deviceC;

  for(int i = 0; i < row1; i++)
  {
      for(int j = 0; j < col1; j++)
          {
              a[i][j] = rand() % RANGE;
              b[i][j] = rand() % RANGE;
          }
  }

  //# ************* GPU MUL *******************// 
  hipMalloc((void **)&deviceA, row1 * col1 * sizeof(int));
  hipMalloc((void **)&deviceB, row2 * col2 * sizeof(int));
  hipMalloc((void **)&deviceC, row1 * col2 * sizeof(int));

  hipMemcpy(deviceA, a, row1 * col1 * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(deviceB, b, row2 * col2 * sizeof(int), hipMemcpyHostToDevice);

  dim3 grid(col2,row1);
  clock_t start = clock();
  matrixMulSharedMemory<<<grid,col1>>>(deviceA, deviceB, deviceC);
  double time = clock() - start;

  hipMemcpy(c, deviceC, row1 * col2 * sizeof(int), hipMemcpyDeviceToHost);
  printf("Time taken (parallel) :: %lf", time / CLOCKS_PER_SEC);

  //# ***************** CPU MUL ****************//
  int actual[row1][col2];
  
  start = clock();
  mulCPU(a, b, actual);
  time = clock() - start;

  printf("\nTime taken (serial) :: %lf", time / CLOCKS_PER_SEC);

  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  return 0;
}