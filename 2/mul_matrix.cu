#include "hip/hip_runtime.h"
%%cu
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>
#define ROWS 128
#define COLS 128
#define RANGE 1000

using namespace std;

//# kernel for matrix mul
__global__ void matrixMul_device(int * a, int * b, int * c){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int sum = 0;
    if(row < ROWS && col < COLS){
        for(int i = 0; i < ROWS; i++){
            sum += a[row * ROWS + i] * b[i * COLS + col];
        }
        __syncthreads();
        c[row * ROWS + col] = sum;
    }
}

//# host helper function
void matrixMul_host(int * a, int * b, int * c){
    dim3 threadsPerBlock(ROWS, ROWS);
    dim3 blocksPerGrid(1, 1);

    if( ROWS * ROWS > 512 ){
        threadsPerBlock.x = 512;
        threadsPerBlock.y = 512;
       
        blocksPerGrid.x = ceil(double(ROWS) / double(threadsPerBlock.x));
        blocksPerGrid.y = ceil(double(COLS) / double(threadsPerBlock.y));
    }

    matrixMul_device<<<blocksPerGrid, threadsPerBlock>>>(a, b, c);
}

int main(){
    int * hostA = (int *) malloc(sizeof(int) * ROWS * COLS);
    int * hostB = (int *) malloc(sizeof(int) * ROWS * COLS);
    int * hostC = (int *) malloc(sizeof(int) * ROWS * COLS);

    int * deviceA , * deviceB, * deviceC;

    for(int i = 0; i < ROWS; i++){
        for(int j = 0; j < COLS; j++){
            hostA[i * COLS + j] = rand() % RANGE;
            hostB[i * COLS + j] = rand() % RANGE;
        }
    }

    //# *************** MUL OPERATION ********************#
    hipMalloc(&deviceA, sizeof(int) * ROWS * COLS);
    hipMalloc(&deviceB, sizeof(int) * ROWS * COLS);
    hipMalloc(&deviceC, sizeof(int) * ROWS * COLS);

    hipMemcpy(deviceA, hostA, sizeof(int) * ROWS * COLS, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, sizeof(int) * ROWS * COLS, hipMemcpyHostToDevice);

    clock_t start = clock();
    matrixMul_host(deviceA, deviceB, deviceC);
    clock_t end = clock();

    hipMemcpy(hostC, deviceC, sizeof(int) * ROWS * COLS, hipMemcpyDeviceToHost);
    printf("Time taken (parallel) :: %lf", (double) (end - start) / CLOCKS_PER_SEC);

    //# ********************** CPU MUL *****************#
    int * actual = (int *) malloc(sizeof(int) * ROWS * COLS);
    int sum;

    start = clock();
    for(int i = 0; i < ROWS; i++){
        for(int j = 0; j < COLS; j++){
            sum = 0;
            for(int k = 0; k < COLS; k++){
                sum += hostA[i * ROWS + k] * hostB[k * ROWS + j];
            }
            actual[i * ROWS + j] = sum;
        }
    }
    end = clock();
    printf("\nTime taken (serial) :: %lf", (double) (end - start) / CLOCKS_PER_SEC);

    //# ******* ERROR CHECK ****************** #
    int error = 0;
    for(int i = 0; i < ROWS * COLS; i++){
        error += abs(actual[i] - hostC[i]);
    }

    printf("\n\nError :: %d", sqrt(error));


    hipFree( deviceA );
    hipFree( deviceB );
    hipFree( deviceC );

    free( hostA );
    free( hostB );
    free( hostC );

    return 0;
} 
