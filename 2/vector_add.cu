#include "hip/hip_runtime.h"
%%cu
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>
#define N 32768
#define RANGE 1000

using namespace std;

//# summation kernel for two vectors
__global__ void add( int *a, int *b, int *c ) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

int main(){
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;

    a = (int*)malloc( N * sizeof(int) );
    b = (int*)malloc( N * sizeof(int) );
    c = (int*)malloc( N * sizeof(int) );

    for (int i=0; i<N; i++) {
        a[i] = rand();
        b[i] = rand();
    }

    //# *************** SUMMATION *********************** #
    hipMalloc( (void**)&dev_a, N * sizeof(int) );
    hipMalloc( (void**)&dev_b, N * sizeof(int) );
    hipMalloc( (void**)&dev_c, N * sizeof(int) );

    hipMemcpy( dev_a, a, N * sizeof(int),hipMemcpyHostToDevice );
    hipMemcpy( dev_b, b, N * sizeof(int),hipMemcpyHostToDevice );

    int numThread = 512;
    int numBlock = ceil( double(N) / double(numThread) );

    clock_t start = clock();
    add<<<numBlock,numThread>>>( dev_a, dev_b, dev_c );
    clock_t end = clock();

    hipMemcpy( c, dev_c, N * sizeof(int),hipMemcpyDeviceToHost );
    printf("Time taken (parallel) :: %lf s", (double) (end - start) / CLOCKS_PER_SEC);

    //# *************** CPU CHECK *********************** #
    int error = 0;

    start = clock();
    for(int i = 0; i < N; i++){
        int diff = (a[i] + b[i]) - c[i];
        error += diff;
    }
    end = clock();
    printf("\nErrors :: %d", error);
    printf("\nTime taken (serial) :: %lf s", (double) (end - start) / CLOCKS_PER_SEC);

    free( a );
    free( b );
    free( c );

    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );

    return 0;
}
