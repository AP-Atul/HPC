#include "hip/hip_runtime.h"
%%cu
#include <bits/stdc++.h>
#include <time.h>
#include <hip/hip_runtime.h>
using namespace std;

int MATRIX = 10000;
int VECTOR = 1000;
int RANGE = 1000;

// # kernel for multiplication
__global__ void multiplication(int * A, int * B, int * C, int MATRIX, int VECTOR){
    int N = MATRIX;
    int M = VECTOR;
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int sum = 0;

    if(index < N){
        for(int i = 0; i < M; i++)
          sum += A[i] * B[(index * M) + i];

        C[index] = sum;
    }
}

// # cpu multiplication
void mulCPU(int * A, int * B, int * C){
    for(int i = 0; i < MATRIX; i++){
	    int sum = 0;
	    for(int j = 0; j < VECTOR; j++){
		    sum += A[j] * B[i * VECTOR + j];
	    }
	    C[i] = sum;
	  }
}

int main(){
    int * hostA = (int *) malloc(VECTOR * sizeof(int));
    int * hostB = (int *) malloc(MATRIX * VECTOR * sizeof(int));
    int * hostC = (int *) malloc(VECTOR * sizeof(int));

    int * deviceA, * deviceB, * deviceC;

    for(int i = 0; i < VECTOR; i++) hostA[i] = rand() % RANGE;
    for(int i = 0; i < MATRIX; i++) for(int j = 0; j < VECTOR; j++) hostB[i * VECTOR + j] = rand() % RANGE;

    //# ******************** MUL OPERATION GPU ********************************
    hipMalloc(&deviceA, VECTOR * sizeof(int));
    hipMalloc(&deviceB, MATRIX * VECTOR * sizeof(int));
    hipMalloc(&deviceC, VECTOR * sizeof(int));

    hipMemcpy(deviceA, hostA, VECTOR * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceB, hostB, MATRIX * VECTOR * sizeof(int), hipMemcpyHostToDevice);

    clock_t start = clock();
    multiplication<<<MATRIX, 1>>>(deviceA, deviceB, deviceC, MATRIX, VECTOR);
    clock_t end = clock();

    hipMemcpy(hostC, deviceC, VECTOR * sizeof(int), hipMemcpyDeviceToHost);
    printf("Time taken (parallel) :: %lf s", (double) (end - start) / CLOCKS_PER_SEC);

    //# ******************** MUL OPERATION CPU ********************************
    int * answer = (int *) malloc(VECTOR * sizeof(int));

    start = clock();
    mulCPU(hostA, hostB, answer);
    end = clock();

    printf("\nTime taken (serial) :: %lf s", (double) (end - start) / CLOCKS_PER_SEC);

    //# ******************** ERROR CHECKING ********************************
    double error = 0;
    for(int i = 0; i < VECTOR; i++){
        if(i == 10) printf("\n%d %d", hostC[i], answer[i]);
        
        error += double(answer[i] - hostC[i]);
    }

    printf("\nError: %lf", error);

    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);

    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}
