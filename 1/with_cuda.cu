#include "hip/hip_runtime.h"
%%cu
#include<iostream>
#include<math.h>
#include<time.h>
#define n 1024

using namespace std;

//# minimum kernel
__global__ void minimum(int *input) {
    int tid = threadIdx.x;
    int step_size = 1;
    int number_of_threads = blockDim.x;
    
    while(number_of_threads>0) {
        if(tid < number_of_threads) {
            int first = tid*step_size*2;
            int second = first + step_size;
            if(input[second] < input[first])
              input[first] = input[second];
        }
        step_size <<= 1;
        number_of_threads >>= 1;
    }
}

//# maximum kernel
__global__ void maximum(int *input) {
    int tid = threadIdx.x;
    int step_size = 1;
    int number_of_threads = blockDim.x;
    
    while(number_of_threads>0) {
        if(tid < number_of_threads) {
            int first = tid*step_size*2;
            int second = first + step_size;
            if(input[second] > input[first])
              input[first] = input[second];
        }
        step_size <<= 1;
        number_of_threads >>= 1;
    }
}

//# summation kernel
__global__ void sum(int *input) {
    const int tid = threadIdx.x;
    int step_size = 1;
    int number_of_threads = blockDim.x;
    
    while(number_of_threads > 0) {
        if(tid < number_of_threads) {
            int first = tid * step_size * 2;
            int second = first + step_size;
            
            input[first] += input[second];
        }
        step_size <<= 1;
        number_of_threads >>= 1;
       
    }
}

//# mean diff kernel
__global__ void mean_diff_sq(float *input, float mean) {
    input[threadIdx.x] -= mean;
    input[threadIdx.x] *= input[threadIdx.x];
}

//# sum of all mean diff kernel
__global__ void sum_floats(float *input) {
    int tid = threadIdx.x;
    int step_size = 1;
    int number_of_threads = blockDim.x;
    
    while(number_of_threads > 0) {
        if(tid < number_of_threads) {
            int first = tid * step_size * 2;
            int second = first + step_size;
            
            input[first] += input[second];
        }
        step_size <<= 1;
        number_of_threads >>= 1;
       
    }
}

//# copy array as float
void copy_int_to_float(float *dest, int *src, int size){
    for(int i=0; i<size; i++)
        dest[i] = float(src[i]);
}


//# driver code
int main() {
    int size = n*sizeof(int); 
    int *arr;
    int *arr_d, result;
    int minCPU = 10001, maxCPU = 0, sumCPU = 0;
   
    arr = (int *)malloc(size);
    for(int i=0; i<n; i++)  {
        int r = rand()%1000;
        if(r == 0)
          r = rand()%1000;
        arr[i] = r;
    }
    
    hipMalloc((void **)&arr_d, size);
    
    //#*************** Minimum operation ****************#
    hipMemcpy(arr_d, arr, size, hipMemcpyHostToDevice);

    clock_t start = clock();
    minimum<<<1,n/2>>>(arr_d);
    clock_t end = clock();

    hipMemcpy(&result, arr_d, sizeof(int), hipMemcpyDeviceToHost);
    
    cout<<"The minimum element is (GPU) :: " << result << endl;
    printf("Time taken (parallel) :: %lf s", (double) (end - start) / CLOCKS_PER_SEC);

    start = clock();
    for(int i = 0; i < n; i++){
         if(arr[i] < minCPU) 
          minCPU = arr[i];
    }
    end = clock();

    cout<<"\n\nThe minimum element is (CPU) "<< minCPU << endl;
    printf("Time taken (serial) :: %lf s", (double) (end - start) / CLOCKS_PER_SEC);
    
       
    //#*************** Maximum operation ****************#
    hipMemcpy(arr_d, arr, size, hipMemcpyHostToDevice);
    
    start = clock();
    maximum<<<1,n/2>>>(arr_d);
    end = clock();
    
    hipMemcpy(&result, arr_d, sizeof(int), hipMemcpyDeviceToHost);
    
    cout << "\n\n\n\nThe maximum element is (GPU) :: " << result << endl;
    printf("Time taken (parallel) :: %lf s", (double) (end - start) / CLOCKS_PER_SEC);

    start = clock();
    for(int i = 0; i < n; i++){
         if(arr[i] > maxCPU) 
          maxCPU = arr[i];
    }
    end = clock();

    cout<<"\n\nThe maximum element is (CPU) "<< maxCPU << endl;
    printf("Time taken (serial) :: %lf s", (double) (end - start) / CLOCKS_PER_SEC);
    
    //#*************** Summation operation ****************#
    hipMemcpy(arr_d, arr, size, hipMemcpyHostToDevice);
    
    start = clock();
    sum<<<1,n/2>>>(arr_d);
    end = clock();
    
    hipMemcpy(&result, arr_d, sizeof(int), hipMemcpyDeviceToHost);
    
    cout << "\n\n\n\nThe summation of elements is (GPU) :: " << result << endl;
    printf("Time taken (parallel) :: %lf s", (double) (end - start) / CLOCKS_PER_SEC);

    start = clock();
    for(int i = 0; i < n; i++){
         sumCPU += arr[i];
    }
    end = clock();

    cout<<"\n\nThe summation of element is (CPU) "<< sumCPU << endl;
    printf("Time taken (serial) :: %lf s", (double) (end - start) / CLOCKS_PER_SEC);
    
    
    //#*************** Average operation ****************#    
    float mean = float(result)/n;
    cout << "\n\n\n\nThe mean/average is " << mean << endl;
    
    
    //#*************** Standard deviation operation ****************#
    float *arr_float;
    float *arr_std, stdValue;
    
    arr_float = (float *)malloc(n*sizeof(float));
    hipMalloc((void **)&arr_std, n*sizeof(float));
    copy_int_to_float(arr_float, arr, n);
    hipMemcpy(arr_std, arr_float, n*sizeof(float), hipMemcpyHostToDevice);
    
    start = clock();
    mean_diff_sq <<<1,n>>>(arr_std, mean);
    sum_floats<<<1,n/2>>>(arr_std);
    end = clock();

    hipMemcpy(&stdValue, arr_std, sizeof(float), hipMemcpyDeviceToHost);
    
    stdValue = stdValue / n;
    stdValue = sqrt(stdValue);

    cout << "\n\nThe standard deviation of element is (GPU) :: " << stdValue << endl;
    printf("Time taken (parallel) :: %lf s", (double) (end - start) / CLOCKS_PER_SEC);

    hipFree(arr_d);
           
    return 0;
}
